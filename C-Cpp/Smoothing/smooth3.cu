#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2012 by J�rn Dinkla, www.dinkla.com, All rights reserved.
 */

#include "smooth3.h"
#include "CAccum.h"

__global__
void smooth_kernel3(const CExtent extent, const uchar4* d_input, uchar4* d_output, const int windowSize) {
	const int x = blockIdx.x * blockDim.x + threadIdx.x;
	const int y = blockIdx.y * blockDim.y + threadIdx.y;
	if (extent.inBounds(x, y)) {
		CAccum acc;
		for (int dy = -windowSize; dy <= windowSize; dy ++) {
			for (int dx = -windowSize; dx <= windowSize; dx ++) {
				const int nx = x + dx;
				const int ny = y + dy;
				if (extent.inBoundsStrict(nx, ny)) {
					acc.add(d_input[extent.index(nx, ny)]);
				}
			}
		}
		d_output[extent.index(x, y)] = acc.avg();
	}
}

void smooth3(const CExecConfig& config, const CExtent& extent, const uchar4* d_input, uchar4* d_output, const int windowSize) {
	smooth_kernel3<<<config.grid,config.threads>>>(extent, d_input, d_output, windowSize);
}

void smooth3(const CExecConfig& config, const CDeviceBuffer<uchar4>& input, const CDeviceBuffer<uchar4>& output, const int windowSize) {
	smooth_kernel3<<<config.grid,config.threads>>>(output, input.getPtr(), output.getPtr(), windowSize);
}
