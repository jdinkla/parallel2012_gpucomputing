#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2012 by J�rn Dinkla, www.dinkla.com, All rights reserved.
 */

#include "CExtent.h"
#include "CAccum.h"
#include "CExecConfig.h"
#include "CPinnedHostBuffer.h"
#include "CDeviceBuffer.h"
#include "CBufferPair.h"

void initialize(uchar4* h_input, const CExtent& extent) {
	for (int y=0; y<extent.height; y++) {
		for (int x=0; x<extent.width; x++) {
			h_input[extent.index(x, y)] = make_uchar4(x, y, 0, x+y);
		}
	}

}
void initialize(uchar4* h_input, const int width, const int height) {
	for (int y=0; y<height; y++) {
		for (int x=0; x<width; x++) {
			h_input[y * width + x] = make_uchar4(x, y, 0, x+y);
		}
	}
}

/*
__device__ inline void add(int4& a, const uchar4& b) {
	a.x += b.x; a.y += b.y; a.z += b.z;
}

__global__ void smooth(const uchar4* d_input, uchar4* d_output, 
			const int width, const int height, const int windowSize) {
	const int x = blockIdx.x * blockDim.x + threadIdx.x;
	const int y = blockIdx.y * blockDim.y + threadIdx.y;
	if (x < width && y < height) {
		...
		d_output[y * width + x] = make_uchar4(a.x/c, a.y/c, a.z/c, 255);
	}
}
*/

__global__ 
void smooth(const CExtent extent, 
			const uchar4* d_input, uchar4* d_output, 
			const int windowSize) {
	const int x = blockIdx.x * blockDim.x + threadIdx.x;
	const int y = blockIdx.y * blockDim.y + threadIdx.y;
	const int z = blockIdx.z * blockDim.z + threadIdx.z;
	if (extent.inBounds(x, y, z)) {
		CAccum acc;
		for (int dz = -windowSize; dz <= windowSize; dz ++) {
			for (int dy = -windowSize; dy <= windowSize; dy ++) {
				for (int dx = -windowSize; dx <= windowSize; dx ++) {
					int nx=x+dx; int ny=y+dy; int nz=z+dz;
					if (extent.inBoundsStrict(nx, ny, nz)) {
						acc.add(d_input[extent.index(nx, ny, nz)]);
					}
				}
			}
		}
		d_output[extent.index(x, y, z)] = acc.avg();
	}
}
	
/*
int main3(int argc, char** argv) {

	CExtent extent(1024, 1024, 1); int windowSize = 1;

	CBufferPair<uchar4> input(extent); input.malloc(); 
	CBufferPair<uchar4> output(extent); output.malloc();

	initialize(input.host); 
	input.host->incVersion(); input.updateDevice();

	CExecConfig config(extent);
	smooth<<<config.grid,config.threads>>>(extent, 
		input.device->getPtr(), output.device->getPtr(), windowSize);

	output.device->incVersion(); output.updateHost();

	input.free(); output.free();

	return 0;
}
 */