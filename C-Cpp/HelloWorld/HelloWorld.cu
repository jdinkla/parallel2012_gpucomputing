/*
 * Copyright (c) 2012 by J�rn Dinkla, www.dinkla.com, All rights reserved.
 */


#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void hello()
{
    int i = threadIdx.x;
	printf("Hello World %i\n", i);
}

int main()
{
	hello<<<1, 3>>>();
	hipDeviceSynchronize();
}
